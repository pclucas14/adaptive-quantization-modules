#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ 
void ChamferDistanceKernel(
	int b,
	int n,
	const float* xyz,
	int m,
	const float* xyz2,
	float* result,
	int* result_i)
{
	const int batch=512;
	__shared__ float buf[batch*3];
	for (int i=blockIdx.x;i<b;i+=gridDim.x){
		for (int k2=0;k2<m;k2+=batch){
			int end_k=min(m,k2+batch)-k2;
			for (int j=threadIdx.x;j<end_k*3;j+=blockDim.x){
				buf[j]=xyz2[(i*m+k2)*3+j];
			}
			__syncthreads();
			for (int j=threadIdx.x+blockIdx.y*blockDim.x;j<n;j+=blockDim.x*gridDim.y){
				float x1=xyz[(i*n+j)*3+0];
				float y1=xyz[(i*n+j)*3+1];
				float z1=xyz[(i*n+j)*3+2];
				int best_i=0;
				float best=0;
				int end_ka=end_k-(end_k&3);
				
				for (int k=0;k<end_ka;k+=1){
					{
						float x2=buf[k*3+0]-x1;
						float y2=buf[k*3+1]-y1;
						float z2=buf[k*3+2]-z1;
						float d=x2*x2+y2*y2+z2*z2;
						if (k==0 || d<best){
							best=d;
							best_i=k+k2;
						}
					}
				}
				
	
				if (k2==0 || result[(i*n+j)]>best){
					result[(i*n+j)]=best;
					result_i[(i*n+j)]=best_i;
				}
			}
			__syncthreads();
		}
	}
}

void ChamferDistanceKernelLauncher(
    const int b, const int n,
    const float* xyz,
    const int m,
    const float* xyz2,
    float* result,
    int* result_i,
    float* result2,
    int* result2_i)
{
	ChamferDistanceKernel<<<dim3(32,16,1),512>>>(b, n, xyz, m, xyz2, result, result_i);
	ChamferDistanceKernel<<<dim3(32,16,1),512>>>(b, m, xyz2, n, xyz, result2, result2_i);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	    printf("error in chamfer distance updateOutput: %s\n", hipGetErrorString(err));
}


